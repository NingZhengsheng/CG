#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "hip/hip_runtime.h"
#include "hipsparse.h"
#include ""
#include <ctime>
#include <fstream>
#include <iostream>
#include <memory.h>
#include <stdio.h>
#include <stdlib.h>

using namespace std;

FILE *f1, *f2, *f3;                     // read data-> A,b,x0
int nx = 360, ny = 180, nz = 38;        // 3D scale
static const int size = 360 * 180 * 38; // size->360*180*38;
static const int NEIGHBOR = 19;         // neighbor->19
double *A, *b, *x;                      // variable
int num_elements = 0;                   // number of elements
int num_rows = 360 * 180 * 38;          // number of rows

int *ptr;
int *col;
double *val;
/*csr ilu0*/
double *val_m;
int *mid_index; //对角线元素index

double *Ap;
double *Ar;
double *r;
double *p;
double alpha = 0;
double *beta;
int iteration = 1;
double threshold = 1e-10; //��ֵ

//回代过程中间变量
double *y;
double *r1;

#define THREADS_PER_BLOCK 512
#define IDX(i, j, k) ((i)*ny * nz + (j)*nz + (k)) // idx of vector
#define IDXA(i, j, k, q)                                                       \
  ((i)*ny * nz * NEIGHBOR + (j)*nz * NEIGHBOR + (k)*NEIGHBOR +                 \
   (q)) // idx of matrix
#define MAX_ITERATION 150
int num_k = 10;
double serialLU = 0;
int main() {

  __host__ void readMatrix();
  __host__ void initData(hipsparseHandle_t handle, hipblasHandle_t handle2,
                         hipsparseMatDescr_t descr, double *d_val, int *d_ptr,
                         int *d_col, double *d_x, double *d_b, double *d_r,
                         double *d_p, double *d_r1);
  __host__ void spmvAp(hipsparseHandle_t handle, hipsparseMatDescr_t descr,
                       double *d_val, int *d_ptr, int *d_col, double *d_p,
                       double *d_Ap);

  __host__ void calAlpha(hipblasHandle_t handle2, double *d_r, double *d_Ap);
  __host__ void calXAndR(hipblasHandle_t handle2, double *d_p, double *d_Ap,
                         double *d_r, double *d_x);

  __host__ void spmvAr(hipsparseHandle_t handle, hipsparseMatDescr_t descr,
                       double *d_val, int *d_ptr, int *d_col, double *d_r1,
                       double *d_Ar);

  __host__ void calBeta(hipblasHandle_t handle2, double *d_Ar, double *d_Ap,
                        double *d_beta);

  __host__ void calPAndAp(hipblasHandle_t handle2, double *d_p, double *d_Ap,
                          double *d_r1, double *d_Ar, double *p_temp,
                          double *Ap_temp);
  __host__ void memFree();

  /*ilu0*/
  __host__ void ilu0(); // ilu0分解
  __host__ void LU();   //三角回代
  // time start
  clock_t start_s, stop_s;
  double duration_s;

  start_s = clock();
  readMatrix();
  stop_s = clock();
  duration_s = (double)(stop_s - start_s) * 1000 / CLOCKS_PER_SEC;
  printf("read time: %.0lf (ms)\n", duration_s);
  ilu0(); // ilu分解

  int *d_ptr;
  int *d_col;
  double *d_val;

  double *d_b;
  double *d_x;
  double *d_Ap;
  double *d_Ar;
  double *d_r;
  double *d_r1;
  double *d_p;
  double *d_beta;
  double *p_temp;
  double *Ap_temp;

  hipMalloc((void **)&d_ptr, sizeof(int) * (num_rows + 1));
  hipMalloc((void **)&d_col, sizeof(int) * (num_elements));
  hipMalloc((void **)&d_val, sizeof(double) * (num_elements));
  hipMalloc((void **)&d_b, sizeof(double) * (num_rows));
  hipMalloc((void **)&d_x, sizeof(double) * (num_rows));
  hipMalloc((void **)&d_Ap, sizeof(double) * (num_rows * num_k));
  hipMalloc((void **)&d_Ar, sizeof(double) * (num_rows));
  hipMalloc((void **)&d_r, sizeof(double) * (num_rows));
  hipMalloc((void **)&d_r1, sizeof(double) * (num_rows));
  hipMalloc((void **)&d_p, sizeof(double) * (num_rows * num_k));
  hipMalloc((void **)&d_beta, sizeof(double) * (num_k));
  hipMalloc((void **)&p_temp, sizeof(double) * (num_rows));
  hipMalloc((void **)&Ap_temp, sizeof(double) * (num_rows));

  hipMemcpy(d_ptr, ptr, sizeof(int) * (num_rows + 1), hipMemcpyHostToDevice);
  hipMemcpy(d_col, col, sizeof(int) * (num_elements), hipMemcpyHostToDevice);
  hipMemcpy(d_val, val, sizeof(double) * (num_elements),
             hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, sizeof(double) * (num_rows), hipMemcpyHostToDevice);
  hipMemcpy(d_x, x, sizeof(double) * (num_rows), hipMemcpyHostToDevice);
  hipMemcpy(d_Ap, Ap, sizeof(double) * (num_rows * num_k),
             hipMemcpyHostToDevice);
  hipMemcpy(d_p, p, sizeof(double) * (num_rows * num_k),
             hipMemcpyHostToDevice);
  hipMemcpy(d_beta, beta, sizeof(double) * (num_k), hipMemcpyHostToDevice);
  hipsparseHandle_t handle;
  hipsparseMatDescr_t descr;
  hipblasHandle_t handle2;

  /* initialize cusparse and cublas library */
  hipsparseStatus_t stat1 = hipsparseCreate(&handle);
  if (stat1 != HIPSPARSE_STATUS_SUCCESS) {
    printf("CUSPARSE initialization failed\n");
    return EXIT_FAILURE;
  }
  hipblasStatus_t stat2 = hipblasCreate(&handle2);
  if (stat2 != HIPBLAS_STATUS_SUCCESS) {
    printf("CUBLAS initialization failed\n");
    return EXIT_FAILURE;
  }
  /* create and setup matrix descriptor */
  hipsparseCreateMatDescr(&descr);
  hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

  int blockNum = (num_rows % THREADS_PER_BLOCK) > 0
                     ? (num_rows / THREADS_PER_BLOCK) + 1
                     : (num_rows / THREADS_PER_BLOCK);

  cout << "blockNum:" << blockNum << endl;
  cout << "***********" << endl;

  initData(handle, handle2, descr, d_val, d_ptr, d_col, d_x, d_b, d_r, d_p,
           d_r1);
  spmvAp(handle, descr, d_val, d_ptr, d_col, d_p, d_Ap);

  hipEvent_t start, stop, LUSTart, LUStop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventCreate(&LUSTart);
  hipEventCreate(&LUStop);
  hipEventRecord(start, 0);
  // cout << "residual:"<<residual() << endl;
  double residual;
  hipblasDnrm2(handle2, num_rows, d_r, 1, &residual);
  cout << "residual:" << residual * residual << endl;
  while (iteration < MAX_ITERATION && residual * residual > threshold) {

    calAlpha(handle2, d_r, d_Ap);
    calXAndR(handle2, d_p, d_Ap, d_r, d_x);
    hipMemcpy(r, d_r, sizeof(double) * num_rows, hipMemcpyDeviceToHost);
    /*serial*/
    hipEventRecord(LUSTart, 0);
    LU();
    hipEventRecord(LUStop, 0);
    hipEventSynchronize(LUStop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, LUSTart, LUStop);
    serialLU += elapsedTime;
    hipMemcpy(d_r1, r1, sizeof(double) * num_rows, hipMemcpyHostToDevice);
    spmvAr(handle, descr, d_val, d_ptr, d_col, d_r1, d_Ar);
    calBeta(handle2, d_Ar, d_Ap, d_beta);
    calPAndAp(handle2, d_p, d_Ap, d_r1, d_Ar, p_temp, Ap_temp);
    hipblasDnrm2(handle2, num_rows, d_r, 1, &residual);
    iteration++;
    cout << "residual:" << residual * residual << endl;
  }
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);

  cout << "***********" << endl;
  cout << "iteration:" << iteration << endl;
  cout << "residual:" << residual * residual << endl;
  cout << "time:" << elapsedTime / 1000 << "s" << endl;
  cout << "LU time:" << serialLU / 1000 << "s" << endl;

  memFree();
  hipFree(d_ptr);
  hipFree(d_val);
  hipFree(d_col);
  hipFree(d_x);
  hipFree(d_b);
  hipFree(d_p);
  hipFree(d_Ap);
  hipFree(d_r);
  hipFree(d_Ar);
  hipFree(d_beta);
  hipFree(p_temp);
  hipFree(Ap_temp);

  hipsparseDestroyMatDescr(descr);
  hipsparseDestroy(handle);
  hipblasDestroy(handle2);
  return 0;
}

/*
 *ilu0分解，分为单位下三角矩阵和上三角矩阵，合并保存到原矩阵空间
 */
void ilu0() {
  int i, j, k;
  int start, end;
  int startk, endk;
  for (i = 1; i < num_rows; i++) {
    start = ptr[i];
    end = ptr[i + 1];
    for (k = start; col[k] < i; k++) {
      val_m[k] /= val_m[mid_index[col[k]]];

      startk = ptr[col[k]];   // col k 行 start
      endk = ptr[col[k] + 1]; // col k 行 end
      for (j = k + 1; j < end; j++) {
        while (col[startk] < col[j] &&
               startk < endk) { // col k行找j列，两种情况退出 找到或没找到
          startk++;
        }
        if (col[startk] == col[j]) { //没找到跳过
          val_m[j] -= val_m[k] * val_m[startk];
        }
      }
    }
  }
}

/*
 *三角回代
 */
__host__ void LU() {
  double temp;
  for (int i = 0; i < num_rows; i++) {
    int start = ptr[i];
    temp = 0;
    for (int j = start; col[j] < i; j++) {
      temp += val_m[j] * y[col[j]];
    }

    y[i] = r[i] - temp;
  }

  for (int i = num_rows - 1; i >= 0; i--) {
    int end = ptr[i + 1];
    temp = 0;
    for (int j = end - 1; col[j] > i; j--) {
      temp += val_m[j] * r1[col[j]];
    }
    r1[i] = (y[i] - temp) / val_m[mid_index[i]];
  }
}

__host__ void memFree() {
  free(ptr);
  free(col);
  free(val);
  hipHostFree(x);
  free(b);
  hipHostFree(Ap);
  hipHostFree(Ar);
  hipHostFree(p);
  hipHostFree(r);
  hipHostFree(r1);
  hipHostFree(beta);
}

__host__ void calPAndAp(hipblasHandle_t handle2, double *d_p, double *d_Ap,
                        double *d_r1, double *d_Ar, double *p_temp,
                        double *Ap_temp) {
  hipblasDcopy(handle2, num_rows, d_r1, 1, p_temp, 1);
  hipblasDcopy(handle2, num_rows, d_Ar, 1, Ap_temp, 1);
  for (int j = 0; j < num_k; j++) {
    hipblasDaxpy(handle2, num_rows, &beta[j], d_p + j * num_rows, 1, p_temp, 1);
    hipblasDaxpy(handle2, num_rows, &beta[j], d_Ap + j * num_rows, 1, Ap_temp,
                1);
  }

  hipblasDcopy(handle2, num_rows, p_temp, 1,
              d_p + (iteration % num_k) * num_rows, 1);
  hipblasDcopy(handle2, num_rows, Ap_temp, 1,
              d_Ap + (iteration % num_k) * num_rows, 1);
}

__host__ void calBeta(hipblasHandle_t handle2, double *d_Ar, double *d_Ap,
                      double *d_beta) {
  int start = iteration > num_k
                  ? ((iteration / num_k) - 1) * num_k + iteration % num_k
                  : 0;
  for (int j = start; j < iteration; j++) {
    double s, m;
    hipblasDdot(handle2, num_rows, d_Ar, 1, d_Ap + (j % num_k) * num_rows, 1,
               &s);
    hipblasDdot(handle2, num_rows, d_Ap + (j % num_k) * num_rows, 1,
               d_Ap + (j % num_k) * num_rows, 1, &m);
    beta[j % num_k] = -1 * (s / m);
  }
}

__host__ void spmvAr(hipsparseHandle_t handle, hipsparseMatDescr_t descr,
                     double *d_val, int *d_ptr, int *d_col, double *d_r1,
                     double *d_Ar) {

  double d1 = 1, d2 = 0;
  hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, num_rows, num_rows,
                 num_elements, &d1, descr, d_val, d_ptr, d_col, d_r1, &d2,
                 d_Ar);
}

__host__ void calXAndR(hipblasHandle_t handle2, double *d_p, double *d_Ap,
                       double *d_r, double *d_x) {
  double alpha2 = -alpha;
  hipblasDaxpy(handle2, num_rows, &alpha,
              d_p + ((iteration - 1) % num_k) * num_rows, 1, d_x, 1);
  hipblasDaxpy(handle2, num_rows, &alpha2,
              d_Ap + ((iteration - 1) % num_k) * num_rows, 1, d_r, 1);
}

__host__ void calAlpha(hipblasHandle_t handle2, double *d_r, double *d_Ap) {
  double s, m;
  hipblasDdot(handle2, num_rows, d_r, 1,
             d_Ap + ((iteration - 1) % num_k) * num_rows, 1, &s);
  hipblasDdot(handle2, num_rows, d_Ap + ((iteration - 1) % num_k) * num_rows, 1,
             d_Ap + ((iteration - 1) % num_k) * num_rows, 1, &m);
  alpha = s / m;
}

__host__ void spmvAp(hipsparseHandle_t handle, hipsparseMatDescr_t descr,
                     double *d_val, int *d_ptr, int *d_col, double *d_p,
                     double *d_Ap) {
  double d1 = 1, d2 = 0;
  hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, num_rows, num_rows,
                 num_elements, &d1, descr, d_val, d_ptr, d_col, d_p, &d2, d_Ap);
}

__host__ void initData(hipsparseHandle_t handle, hipblasHandle_t handle2,
                       hipsparseMatDescr_t descr, double *d_val, int *d_ptr,
                       int *d_col, double *d_x, double *d_b, double *d_r,
                       double *d_p, double *d_r1) {
  // int blockNum = (num_rows % THREADS_PER_BLOCK) > 0 ? (num_rows /
  // THREADS_PER_BLOCK) + 1 : (num_rows / THREADS_PER_BLOCK);
  double d2 = 1, d3 = -1;
  hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, num_rows, num_rows,
                 num_elements, &d3, descr, d_val, d_ptr, d_col, d_x, &d2, d_b);
  hipblasDcopy(handle2, num_rows, d_b, 1, d_r, 1);
  hipMemcpy(r, d_r, sizeof(double) * num_rows, hipMemcpyDeviceToHost);
  LU();
  hipMemcpy(d_r1, r1, sizeof(double) * num_rows, hipMemcpyHostToDevice);
  hipblasDcopy(handle2, num_rows, d_r1, 1, d_p, 1);
  /*
  hipMemcpy(r, d_r, sizeof(double) * num_rows, hipMemcpyDeviceToHost);
  hipMemcpy(p, d_p, sizeof(double) * num_rows * num_k, hipMemcpyDeviceToHost);
  cout << r[3] << endl;
  cout << p[3] << endl;
  */
}

void readMatrix() {
  A = (double *)malloc(sizeof(double) * size * NEIGHBOR);
  b = (double *)malloc(sizeof(double) * size);
  x = (double *)malloc(sizeof(double) * size);
  f1 = fopen("case/A", "rb");
  fread(A, sizeof(double), size * NEIGHBOR, f1);
  fclose(f1);
  f2 = fopen("case/b", "rb");
  fread(b, sizeof(double), size, f2);
  fclose(f2);
  f3 = fopen("case/x0", "rb");
  fread(x, sizeof(double), size, f3);
  fclose(f3);
  ptr = (int *)malloc(sizeof(int) * (size + 1));
  col = (int *)malloc(sizeof(int) * size * NEIGHBOR);
  val = (double *)malloc(sizeof(double) * size * NEIGHBOR);
  mid_index = (int *)malloc(sizeof(int) * num_rows); //对角线元素index

  Ap = (double *)malloc(sizeof(double) * num_rows * num_k);
  Ar = (double *)malloc(sizeof(double) * num_rows);
  r = (double *)malloc(sizeof(double) * num_rows);
  r1 = (double *)malloc(sizeof(double) * num_rows); // ilu variable
  y = (double *)malloc(sizeof(double) * num_rows);  // ilu variable
  p = (double *)malloc(sizeof(double) * num_rows * num_k);
  if (p == NULL)
    printf("memory error\n");
  beta = (double *)malloc(sizeof(double) * num_k);
  memset(Ap, 0, num_rows * num_k * sizeof(double));
  memset(p, 0, num_rows * num_k * sizeof(double));
  memset(beta, 0, num_k * sizeof(double));

  ptr[0] = 0;
  int index = 0;

  int i, j, k; // nx,ny,nz
  //#pragma omp parallel private(i,j,k,index)
  {
    //#pragma omp for
    for (i = 0; i < nx; i++)
      for (j = 0; j < ny; j++)
        for (k = 0; k < nz; k++) {
          col[index] = IDX(i, j, k);
          val[index++] = A[IDXA(i, j, k, 0)];

          if (i > 0)
            col[index] = IDX(i - 1, j, k);
          else
            col[index] = IDX(nx - 1, j, k);
          val[index++] = A[IDXA(i, j, k, 1)];

          if (i < nx - 1)
            col[index] = IDX(i + 1, j, k);
          else
            col[index] = IDX(0, j, k);
          val[index++] = A[IDXA(i, j, k, 2)];

          if (j > 0)
            col[index] = IDX(i, j - 1, k);
          else
            col[index] = IDX((i + nx / 2) % nx, j, k);
          val[index++] = A[IDXA(i, j, k, 3)];

          if (j < ny - 1)
            col[index] = IDX(i, j + 1, k);
          else
            col[index] = IDX((i + nx / 2) % nx, j, k);
          val[index++] = A[IDXA(i, j, k, 4)];

          if (i < nx - 1 && j < ny - 1)
            col[index] = IDX(i + 1, j + 1, k);
          else if (i < nx - 1 && j == ny - 1)
            col[index] = IDX((i + 1 + nx / 2) % nx, j, k);
          else if (i == nx - 1 && j < ny - 1)
            col[index] = IDX(0, j + 1, k);
          else
            col[index] = IDX(nx / 2, j, k);
          val[index++] = A[IDXA(i, j, k, 5)];

          if (i < nx - 1 && j > 0)
            col[index] = IDX(i + 1, j - 1, k);
          else if (i < nx - 1 && j == 0)
            col[index] = IDX((i + 1 + nx / 2) % nx, j, k);
          else if (i == nx - 1 && j > 0)
            col[index] = IDX(0, j - 1, k);
          else
            col[index] = IDX(nx / 2, j, k);
          val[index++] = A[IDXA(i, j, k, 6)];

          if (i > 0 && j > 0)
            col[index] = IDX(i - 1, j - 1, k);
          else if (i > 0 && j == 0)
            col[index] = IDX((i - 1 + nx / 2) % nx, j, k);
          else if (i == 0 && j > 0)
            col[index] = IDX(nx - 1, j - 1, k);
          else
            col[index] = IDX((nx - 1 + nx / 2) % nx, j, k);
          val[index++] = A[IDXA(i, j, k, 7)];

          if (i > 0 && j < ny - 1)
            col[index] = IDX(i - 1, j + 1, k);
          else if (i > 0 && j == ny - 1)
            col[index] = IDX((i - 1 + nx / 2) % nx, j, k);
          else if (i == 0 && j < ny - 1)
            col[index] = IDX(nx - 1, j + 1, k);
          else
            col[index] = IDX((nx - 1 + nx / 2) % nx, j, k);
          val[index++] = A[IDXA(i, j, k, 8)];

          if (k > 0)
            col[index] = IDX(i, j, k - 1), val[index++] = A[IDXA(i, j, k, 9)];

          if (k > 0 && i > 0)
            col[index] = IDX(i - 1, j, k - 1),
            val[index++] = A[IDXA(i, j, k, 10)];
          else if (k > 0 && i == 0)
            col[index] = IDX(nx - 1, j, k - 1),
            val[index++] = A[IDXA(i, j, k, 10)];

          if (k > 0 && i < nx - 1)
            col[index] = IDX(i + 1, j, k - 1),
            val[index++] = A[IDXA(i, j, k, 11)];
          else if (k > 0 && i == nx - 1)
            col[index] = IDX(0, j, k - 1), val[index++] = A[IDXA(i, j, k, 11)];

          if (k > 0 && j > 0)
            col[index] = IDX(i, j - 1, k - 1),
            val[index++] = A[IDXA(i, j, k, 12)];
          else if (k > 0 && j == 0)
            col[index] = IDX((i + nx / 2) % nx, j, k - 1),
            val[index++] = A[IDXA(i, j, k, 12)];

          if (k > 0 && j < ny - 1)
            col[index] = IDX(i, j + 1, k - 1),
            val[index++] = A[IDXA(i, j, k, 13)];
          else if (k > 0 && j == ny - 1)
            col[index] = IDX((i + nx / 2) % nx, j, k - 1),
            val[index++] = A[IDXA(i, j, k, 13)];

          if (k < nz - 1)
            col[index] = IDX(i, j, k + 1), val[index++] = A[IDXA(i, j, k, 14)];

          if (k < nz - 1 && i > 0)
            col[index] = IDX(i - 1, j, k + 1),
            val[index++] = A[IDXA(i, j, k, 15)];
          else if (k < nz - 1 && i == 0)
            col[index] = IDX(nx - 1, j, k + 1),
            val[index++] = A[IDXA(i, j, k, 15)];

          if (k < nz - 1 && i < nx - 1)
            col[index] = IDX(i + 1, j, k + 1),
            val[index++] = A[IDXA(i, j, k, 16)];
          else if (k < nz - 1 && i == nx - 1)
            col[index] = IDX(0, j, k + 1), val[index++] = A[IDXA(i, j, k, 16)];

          if (k < nz - 1 && j > 0)
            col[index] = IDX(i, j - 1, k + 1),
            val[index++] = A[IDXA(i, j, k, 17)];
          else if (k < nz - 1 && j == 0)
            col[index] = IDX((i + nx / 2) % nx, j, k + 1),
            val[index++] = A[IDXA(i, j, k, 17)];

          if (k < nz - 1 && j < ny - 1)
            col[index] = IDX(i, j + 1, k + 1),
            val[index++] = A[IDXA(i, j, k, 18)];
          else if (k < nz - 1 && j == ny - 1)
            col[index] = IDX((i + nx / 2) % nx, j, k + 1),
            val[index++] = A[IDXA(i, j, k, 18)];

          ptr[IDX(i, j, k) + 1] = index;
        }
  }

  // sort
  for (int i = 0; i < num_rows; i++) {
    int start = ptr[i];
    int end = ptr[i + 1];
    double temp_d;
    int temp_i;
    for (int j = start; j < end - 1; j++) {

      for (int k = start; k < end - 1 - j + start; k++) {

        if (col[k] > col[k + 1]) {
          temp_d = val[k + 1];
          temp_i = col[k + 1];
          val[k + 1] = val[k];
          col[k + 1] = col[k];
          val[k] = temp_d;
          col[k] = temp_i;
        }
      }
    }
  }
  //找每行对角中间元素的index
  for (int i = 0; i < num_rows; i++) {
    int start = ptr[i];
    int end = ptr[i + 1];
    for (int j = start; j < end; j++) {
      if (col[j] == i)
        mid_index[i] = j;
    }
  }

  free(A); // free space
  num_elements = index;
  /*csr ilu0*/
  val_m = (double *)malloc(sizeof(double) * size * NEIGHBOR);
  memcpy(val_m, val, sizeof(double) * size * NEIGHBOR); // cpy val_m
}
